#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <random>

#define DATA_SIZE (1024 * 1024)

// DSMEM histogram kernel
__global__ void clusterHist_kernel(int *bins, const int nbins, const int bins_per_block, const int *__restrict__ input,
                                   size_t array_size)
{
  extern __shared__ int smem[];
  namespace cg = cooperative_groups;
  int tid = cg::this_grid().thread_rank(); // Local thread index

  // Cluster initialization, size and calculating local bin offsets.
  cg::cluster_group cluster = cg::this_cluster(); 
  unsigned int clusterBlockRank = cluster.block_rank(); // Rank of the current block in the cluster
  int cluster_size = cluster.dim_blocks().x; // Number of blocks in the cluster dimension x

  //Initialize shared memory histogram to zeros
  for (int i = threadIdx.x; i < bins_per_block; i += blockDim.x)
  {
    smem[i] = 0; 
  }

  // cluster synchronization ensures that shared memory is initialized to zero in
  // all thread blocks in the cluster. It also ensures that all thread blocks
  // have started executing and they exist concurrently.
  cluster.sync();

  for (int i = tid; i < array_size; i += blockDim.x * gridDim.x)
  {
    int ldata = input[i];

    //Find the right histogram bin.
    int binid = ldata;
    if (ldata < 0)
      binid = 0;
    else if (ldata >= nbins)
      binid = nbins - 1;

    //Find destination block rank and offset for computing
    //distributed shared memory histogram
    int dst_block_rank = (int)(binid / bins_per_block);
    int dst_offset = binid % bins_per_block;

    //Pointer to target block shared memory
    int *dst_smem = cluster.map_shared_rank(smem, dst_block_rank);

    //Perform atomic update of the histogram bin
    atomicAdd(dst_smem + dst_offset, 1);
  }

  // cluster synchronization is required to ensure all distributed shared
  // memory operations are completed and no thread block exits while
  // other thread blocks are still accessing distributed shared memory
  cluster.sync();

  // Perform global memory histogram, using the local distributed memory histogram
  int *lbins = bins + cluster.block_rank() * bins_per_block;
  for (int i = threadIdx.x; i < bins_per_block; i += blockDim.x)
  {
    atomicAdd(&lbins[i], smem[i]);
  }
}

// Non-DSM histogram kernel
__global__ void histogram_kernel(int *bins, const int nbins, const int *__restrict__ input, size_t array_size)
{
    extern __shared__ int smem[];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Initialize shared memory
    for (int i = threadIdx.x; i < nbins; i += blockDim.x) {
        smem[i] = 0;
    }
    __syncthreads();
    
    // Process data
    for (int i = tid; i < array_size; i += blockDim.x * gridDim.x) {
        int ldata = input[i];
        int binid = (ldata < 0) ? 0 : ((ldata >= nbins) ? nbins - 1 : ldata);
        atomicAdd(&smem[binid], 1);
    }
    __syncthreads();
    
    // Write to global memory
    for (int i = threadIdx.x; i < nbins; i += blockDim.x) {
        atomicAdd(&bins[i], smem[i]);
    }
}

void generate_data(std::vector<int>& data, int bins) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(0, bins * 4);
    for (int& val : data) {
        val = dis(gen);
    }
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <bin_size> <cluster_size>" << std::endl;
        return 1;
    }
    
    int bin_size = std::atoi(argv[1]);
    int cluster_size = std::atoi(argv[2]);
    
    // Print shared memory size per SM
    int device = 0;
    hipGetDevice(&device);
    hipDeviceProp_t prop{};
    hipGetDeviceProperties(&prop, device);
    std::cout << "Shared memory per SM (KB): " << prop.sharedMemPerMultiprocessor / 1024 << std::endl;
    std::cout << "Bin size(KB): " << bin_size * sizeof(int) / 1024 << std::endl;
    
    // Generate test data
    std::vector<int> h_data(DATA_SIZE);
    generate_data(h_data, bin_size);
    
    // Allocate GPU memory
    int *d_data, *d_histogram;
    hipMalloc(&d_data, DATA_SIZE * sizeof(int));
    hipMalloc(&d_histogram, bin_size * sizeof(int));
    hipMemcpy(d_data, h_data.data(), DATA_SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_histogram, 0, bin_size * sizeof(int));
    
    // Create events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    if (cluster_size == 1) {
        // Non-DSM case
        int threads_per_block = 1024;
        int total_blocks = std::min(32, (DATA_SIZE + threads_per_block - 1) / threads_per_block);
        size_t smem_size = bin_size * sizeof(int);
        size_t array_size = DATA_SIZE;
        // Launch kernel based on cluster size
        hipEventRecord(start);
        histogram_kernel<<<total_blocks, threads_per_block, smem_size>>>(d_histogram, bin_size, d_data, array_size);
    } else {
        // DSM usage
        int threads_per_block = 1024;
        int bins_per_block = (bin_size + cluster_size - 1) / cluster_size;
        size_t smem_size = bins_per_block * sizeof(int);
        std::cout << "Bins per block: " << bins_per_block << std::endl;
        std::cout << "SMEM size: " << smem_size / 1024 << " KB" << std::endl;
        size_t array_size = DATA_SIZE;
        
        hipLaunchConfig_t config = {0};
        config.gridDim = dim3(cluster_size, 1, 1);
        config.blockDim = dim3(threads_per_block, 1, 1);
        config.dynamicSmemBytes = smem_size;
        
        hipLaunchAttribute attr; 
        attr.id = cudaLaunchAttributeClusterDimension;
        attr.val.clusterDim.x = cluster_size; // number of blocks in the cluster
        attr.val.clusterDim.y = 1; 
        attr.val.clusterDim.z = 1;
        
        // Set cluster dimension attribute for the kernel
        hipFuncSetAttribute(reinterpret_cast<const void*>(clusterHist_kernel), cudaFuncAttributeRequiredClusterWidth, cluster_size); 
        hipFuncSetAttribute(reinterpret_cast<const void*>(clusterHist_kernel), cudaFuncAttributeRequiredClusterHeight, 1);
        hipFuncSetAttribute(reinterpret_cast<const void*>(clusterHist_kernel), cudaFuncAttributeRequiredClusterDepth, 1);
        // Launch kernel based on cluster size
        hipEventRecord(start);
        clusterHist_kernel<<<config.gridDim, config.blockDim, config.dynamicSmemBytes>>>(d_histogram, bin_size, bins_per_block, d_data, array_size);

    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    
    std::cout << bin_size << " " << cluster_size << " " << ms << std::endl;
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_data);
    hipFree(d_histogram);
    
    return 0;
}